
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <sys/time.h>
#include <string.h>


/*

Row wise multiplication algorithm implemented in parallel. Accepts arbitrary numbers of equivalent size.
Both times will be printed to see the benefits of paralle computation for the large integer and answeres
will be verified based upon comparison of both algorithms (pass/fail). Regular row wise multiplication is
used and the carries for each product multiplication is dealt with by a sequential carry adder. The
multiplication algorithm is beneficially in the use of RSA encryption or Diffie Hellman key exchange. The algorithm could be implemented in other applications were large amounts of multiplication can be used in parallel to reduce computation time.

*/
__global__ void get_products(unsigned char a[], unsigned char b[], unsigned int accumulator[], unsigned int n);

int main(int argc, char *argv[]) {	

	if (argc != 3) {
		printf("usage: ./a.out N ThreadsPerBlock\n");
		exit(1);
	}
	printf("Version1, n = %s, threads = %s\n", argv[1], argv[2]);

	unsigned int n = atoi(argv[1]);
	unsigned int threads = atoi(argv[2]);

	unsigned char *p =  (unsigned char *) malloc(n);
	unsigned char *q =  (unsigned char *) malloc(n);


	//replace with the ability to read in file
	int t = 0;
	unsigned char hex;
	while(t < n) {
		hex = (unsigned char) (rand() % 255) + 1;
		p[t] = hex;
		t++;
	}

	t = 0;
	while(t < n) {
		hex = (unsigned char) (rand() % 255) + 1;
		q[t] = hex;
		t++;
	}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Row wise GPU version
////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	unsigned char *d_A;
	hipMalloc(&d_A, n);
	unsigned char *d_B;
	hipMalloc(&d_B, n);
	unsigned int *d_C;
	hipMalloc(&d_C, 2*n*sizeof(unsigned int));

	hipMemcpy(d_A, p, n, hipMemcpyHostToDevice); 
	hipMemcpy(d_B, q, n, hipMemcpyHostToDevice);
	hipMemset(d_C, 0, 2*n*sizeof(unsigned int));		

	dim3 blocksPerGrid(n/threads);
	dim3 threadsPerBlock(threads);

	hipError_t error;
	hipEvent_t start;

	error = hipEventCreate(&start);
	if(error != hipSuccess)
		printf("error\n");
	
	hipEvent_t stop;
	error = hipEventCreate(&stop);
	if(error != hipSuccess)
		printf("error\n");

	error = hipEventRecord(start, NULL);


	//call kernel to multiply a * b = c where a and b are of size n
	get_products<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, n);

	//compute final answer with sequential adder
	unsigned char *final = (unsigned char *) malloc(2*n);
	memset(final, 0, 2*n);
	unsigned int *transfer = (unsigned int *) malloc(2*n*sizeof(unsigned int));

	//copy result of multiplication to cpu copy to calculate carries
	hipMemcpy(transfer, d_C, 2*n*sizeof(unsigned int), hipMemcpyDeviceToHost);
	unsigned int index = 0;
	while(index < 2*n) {
		//cast to character and add to index of final result
		final[index] = (unsigned char) transfer[index];

		//collect the other three bytes and add to the next sequential 
		//integer index
		transfer[index + 1] += (unsigned int) (transfer[index]>>8);
		index++;
	}



	error = hipEventRecord(stop, NULL);

	error = hipEventSynchronize(stop);

	if(error != hipSuccess)
		printf("error\n");

	float msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);// return is miliseconds

	printf("GPU time: %.6f\n", msecTotal / 1000);


///////////////////////////////////////////////////////////////////////////////////////////////////
// Row Wise CPU for time comparison
//////////////////////////////////////////////////////////////////////////////////////////////////
	unsigned char *cpu_result = (unsigned char *) malloc(2*n);
	memset(cpu_result, 0, 2*n);

	unsigned int multiplicand_position;
	struct timeval cpu_start, cpu_end;
	struct timezone tzp;

	gettimeofday(&cpu_start, &tzp);

	//loop through n rows of products
	for(multiplicand_position = 0; multiplicand_position < n; multiplicand_position++) {

		unsigned int result_position = multiplicand_position;
		unsigned char result_carry = 0;
		unsigned short cpu_product = 0;
		unsigned int multiplier_position = 0;
		unsigned short cpu_sum;

		unsigned int loop = 0;
		//loop through n multipliers
		while(loop < n) {

			//calculate the product of ch * ch
			unsigned short cpu_sum;
			cpu_product = p[multiplier_position] * q[multiplicand_position];

			multiplier_position++;

			//calculate the sum of previous carry, current result index, and current product
			cpu_sum = (cpu_result[result_position] + (cpu_product<<8>>8)  + result_carry);

			//shift carry bits from upper half of short sum
			result_carry = (cpu_sum >> 8);

			//update current indexs result
			cpu_result[result_position] = cpu_sum;

			result_position++;
			loop++;
		}

		//compute final carry of last index from each row
		cpu_sum = (cpu_result[result_position] + result_carry);
		cpu_result[result_position] = cpu_sum;

		//update carry for those rows which are not equal to n
		result_carry = (cpu_sum >> 8);
		cpu_result[result_position+ 1] += result_carry;

}

	gettimeofday(&cpu_end, &tzp); // return is in microseconds
	printf("CPU time: %.6f\n", (cpu_end.tv_sec - cpu_start.tv_sec) + (cpu_end.tv_usec - cpu_start.tv_usec) / 1000000.0);

        unsigned int err = 0;
        unsigned int g = 0;
	
	//compare for finding error in the result of cpu vs. gpu
	while(g<2*n){
                if(final[g] != cpu_result[g]) {
                        err++;
                }
                g++;
        }

        if(err == 0)
                printf("PASS\n");

        else
                printf("FAIL\n");
	
	//free memory
	free(p);
	free(q);
	free(cpu_result);
	free(final);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

return 0;
}

//each thread will compute a complete row of products where the index of the kernel array is the multiplicand
//for the specific threads multiplicand. The thread will loop through the other multilpiers to calculate a
//row of products. Atomically add to assure that data is not missed or overwritten.
__global__ void get_products(unsigned char a[], unsigned char b[], unsigned int accumulator[], unsigned int n) {

	int multiplier = 0;
	unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;// kernel index
	unsigned int multiplicand = index;

	//atomic add only uses integers so product will only store a short becuase atomic add integer
	//cannot be casted to a short
	unsigned int product = 0;

	//loop through multipliers and find products
	while(multiplier < n) {
	
		//compute ch * ch and produce a short
		product = (unsigned int) a[multiplier] * b[multiplicand];

		//add the first character to the respective result index
		atomicAdd(&accumulator[multiplier + index], product<<24>>24);
		
		//add the second character to the respective result index
		atomicAdd(&accumulator[multiplier + index + 1], product>>8);
		multiplier++;
	}
return;
}
